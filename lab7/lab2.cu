
#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h> 

__global__ void calPi(double * pi_D)
	{
	int t_rank = (blockIdx.x*blockDim.x) + threadIdx.x ;
	int x = 2 + (2 * t_rank);
	double y = ((double)4/x) * (double)1/(x+1) * (double)1/(x+2);

	if(t_rank % 2 != 0)
		y = -y;

	pi_D[t_rank] = y;
	}

int main()
	{
	printf("pi calculate...\n");

	int thread_size = 500, block_size = 5;
	double *pi_D;
	double *pi_H;

	pi_H = (double*) malloc(sizeof(double) *thread_size*block_size);
	hipMalloc( (void **)&pi_D, sizeof(double)*thread_size*block_size);

	calPi<<<block_size,thread_size>>>(pi_D);

	hipMemcpy(pi_H, pi_D, thread_size*block_size * sizeof(double), hipMemcpyDeviceToHost);
	hipFree(pi_D);

	double pi = 3;
	for(int i = 0 ; i < thread_size*block_size ; i++)
		pi = (double) pi + pi_H[i];

	/* Change double to string, prevent it round the decimal */
	char result[12];
	sprintf(result, "%.11lf", pi);
	result[strlen(result)-1] = '\0';
	printf("calculated pi = %s \n",result);
	}